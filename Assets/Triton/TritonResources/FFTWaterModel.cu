
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <hip/hip_math_constants.h>

__global__ void CreateTextures(const float *H, const float *chopX, const float *chopZ, float * displacement, float * slopeFoam, float *foam,
                               uint2 dim, float2 twoCellsSize, float chopScale, float dt)
{
    int2 gridPos;
    gridPos.x = blockIdx.x*blockDim.x + threadIdx.x;
    gridPos.y =  blockIdx.y*blockDim.y + threadIdx.y;

    int idx = gridPos.y * dim.x + gridPos.x;
    int imageIdx = gridPos.y * dim.x * 4 + gridPos.x * 4;

    displacement[imageIdx] = chopX[idx] * chopScale;
    displacement[imageIdx+1] = chopZ[idx] * chopScale;
    displacement[imageIdx+2] = H[idx];
    displacement[imageIdx+3] = 1.0f;

    int prevX = gridPos.x > 0 ? gridPos.x-1 : dim.x-1;
    int nextX = gridPos.x < dim.x-1 ? gridPos.x + 1 : 0;
    int prevY = gridPos.y > 0 ? gridPos.y - 1 : dim.y-1;
    int nextY = gridPos.y < dim.y - 1 ? gridPos.y + 1 : 0;

    float xWidth = twoCellsSize.x + chopX[gridPos.y * dim.x + nextX] - chopX[gridPos.y * dim.x + prevX];
    float yDepth = twoCellsSize.y + chopZ[nextY * dim.x + gridPos.x] - chopZ[prevY * dim.x + gridPos.x];
    float xDelta = (H[gridPos.y * dim.x + nextX] - H[gridPos.y * dim.x + prevX]);
    float yDelta = (H[nextY * dim.x + gridPos.x] - H[prevY * dim.x + gridPos.x]);
    float dx = xDelta / xWidth;
    float dy = yDelta / yDepth;

    slopeFoam[imageIdx] = dx;
    slopeFoam[imageIdx+1] = dy;

    // Rate at which x displacement changes if x is constant
    float sxx = (chopX[nextY * dim.x + gridPos.x] - chopX[prevY * dim.x + gridPos.x]) / yDepth;
    // Rate at which y displacement changes if x is constant
    float syx = (chopZ[nextY * dim.x + gridPos.x] - chopZ[prevY * dim.x + gridPos.x]) / yDepth;
    // Rate at which y displacement changes if y is constant
    float syy = (chopZ[gridPos.y * dim.x + nextX] - chopZ[gridPos.y * dim.x + prevX]) / xWidth;
    // Rate at which x displacement changes if y is constant
    float sxy = (chopX[gridPos.y * dim.x + nextX] - chopX[gridPos.y * dim.x + prevX]) / xWidth;

    float Jxx = 1.0 + chopScale * sxx;
    float Jyy = 1.0 + chopScale * syy;
    float Jxy = chopScale * sxy;
    float Jyx = chopScale * syx;

    float J = Jxx * Jyy - Jxy * Jyx;

    float spray = 1.0f - J;

    slopeFoam[imageIdx+2] = spray;
    slopeFoam[imageIdx+3] = 1.0f;

    foam[idx] = spray;
}

__global__ void ProcessWater(const float *H0, const float *omega, float *H, float *chopX, float *chopZ, float time, uint2 inDim, uint2 outDim, float2 size)
{
    int2 gridPos;
    gridPos.x = blockIdx.x * blockDim.x + threadIdx.x;
    gridPos.y = blockIdx.y * blockDim.y + threadIdx.y;

    if (gridPos.x >= outDim.x) return;
    if (gridPos.y >= outDim.y) return;

    int h0idx = gridPos.y*(inDim.x + 1)*2 + gridPos.x * 2;
    float2 h0;
    h0.x = H0[h0idx];
    h0.y = H0[h0idx + 1];

    int h0NegKIdx = (inDim.y - gridPos.y) * (inDim.x + 1) * 2 + (inDim.x - gridPos.x) * 2;
    float2 h0NegKConj;
    h0NegKConj.x = H0[h0NegKIdx];
    h0NegKConj.y = H0[h0NegKIdx+1] * -1;

    uint2 half = make_uint2(inDim.x / 2, inDim.y / 2);
    float2 kPos;
    kPos.x = (float)gridPos.x - (float)half.x;
    kPos.y = (float)gridPos.y - (float)half.y;

    const float TWOPI = 3.14159265f * 2.0f;
    float2 K = make_float2((TWOPI * kPos.x) / size.x , (TWOPI * kPos.y) / size.y);

    float wk = omega[inDim.x * gridPos.y + gridPos.x];
    float wkt = wk * time;
    float cwkt = cosf(wkt);
    float swkt = sinf(wkt);

    float2 term1, term2;
    term1.x = h0.x * cwkt - h0.y * swkt;
    term1.y = h0.x * swkt + h0.y * cwkt;
    term2.x = h0NegKConj.x * cwkt - h0NegKConj.y * -swkt;
    term2.y = h0NegKConj.x * -swkt + h0NegKConj.y * cwkt;

    float2 Htilde = make_float2(term1.x + term2.x, term1.y + term2.y);

    int outIdx = gridPos.y * outDim.x * 2 + gridPos.x * 2;
    H[outIdx] = Htilde.x;
    H[outIdx+1] = Htilde.y;

    if (  (K.x * K.x + K.y * K.y) > 0) {
        float invLen = 1.0f / sqrtf(K.x*K.x + K.y*K.y);
        float2 chopImg = make_float2(K.x * invLen, K.y * invLen);
        float2 cX, cZ;
        cX.x = -(chopImg.x * Htilde.y);
        cX.y = (chopImg.x * Htilde.x);
        cZ.x = -(chopImg.y * Htilde.y);
        cZ.y = (chopImg.y * Htilde.x);

        chopX[outIdx] = cX.x;
        chopX[outIdx+1] = cX.y;
        chopZ[outIdx] = cZ.x;
        chopZ[outIdx+1] = cZ.y;
    } else {
        chopX[outIdx] = chopX[outIdx+1] = chopZ[outIdx] = chopZ[outIdx+1] = 0.0f;
    }
}

//Round a / b to nearest higher integer value
int cuda_iDivUp(int a, int b)
{
    return (a + (b - 1)) / b;
}

__global__ void flipSignsKernel(float* h, unsigned int width, unsigned int height)
{
    unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;
    unsigned int i = y*width+x;

    float signs[2] = {1, -1};
    h[i] *= signs[x+y & 1];
}

extern "C" void cudaFlipSignsKernel(  float* hptr,
                                      unsigned int width, unsigned int height)
{
    dim3 block(16, 16, 1);
    dim3 grid2(cuda_iDivUp(width, block.x), cuda_iDivUp(height, block.y), 1);
    flipSignsKernel<<<grid2, block>>>(hptr, width, height);
}

extern "C" void cudaCreateTextures(const float *H, const float *chopX, const float *chopZ, unsigned int width, unsigned int height, float *displacement, float *slopeFoam, float *foam, uint2 dim, float2 twoCellsSize,
                                   float chopScale, float dt)
{
    dim3 block(16, 16, 1);
    dim3 grid2(cuda_iDivUp(width, block.x), cuda_iDivUp(height, block.y), 1);
    CreateTextures<<<grid2, block>>>(H, chopX, chopZ, displacement, slopeFoam, foam, dim, twoCellsSize, chopScale, dt);
}

extern "C" void cudaProcessWater(const float *H0, const float *omega, float *H, float *chopX, float *chopZ, float t, uint2 inDim, uint2 outDim, float2 size)
{
    dim3 block(16, 16, 1);
    dim3 grid2(cuda_iDivUp(outDim.x, block.x), cuda_iDivUp(outDim.y, block.y), 1);
    ProcessWater<<<grid2, block>>>(H0, omega, H, chopX, chopZ, t, inDim, outDim, size);
}
